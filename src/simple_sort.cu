
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__
void sortArray(int* array, int count) {
	int idx = (blockIdx.x * blockDim.x * 2) + threadIdx.x;
	if (idx >= count) { return; }
}

void printArray(int* array, int count) {
	for (int x = 0; x < count; x++) {
		printf("%d ", array[x]);
		if (x % 15 == 0) { printf("\n"); }
	}
}

int main(void) {
	printf("\n");

	srand(time(NULL));

	int count = 5000;
	dim3 block(32);
	dim3 grid((count + block.x - 1) / block.x);

	int* host_array = (int*)malloc(count*sizeof(int));
	for (int x = 0; x < count; x++) { host_array[x] = rand() % count; }

	int* device_array;
	hipMalloc((int**)&device_array, count*sizeof(int));
	hipMemcpy(device_array, host_array, count*sizeof(int), hipMemcpyHostToDevice);

	sortArray<<<grid,block>>>(device_array, count);
	hipDeviceSynchronize();

	hipMemcpy(host_array, device_array, count*sizeof(int), hipMemcpyDeviceToHost);
	printArray(host_array, count);

	printf("\n");
	return 0;
}