#include "hip/hip_runtime.h"
#include <stdio.h>
#include "analysis.h"

__device__ float globalData;

__global__ 
void changeGlobalData() {
	globalData += 1.00f;
}

__global__
void updateArray(int* array, int count) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < count) {
		array[idx] *= 2.00f;
	}
}

__global__
void printDeviceMemoryArray(int* array, int count) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < count) {
		printf("%d ", array[idx]);
	}
}

void printArray(int* array, int count) {
	for (int x = 0; x < count; x++) {
		printf("%d ", array[x]);
	}
	printf("\n");
}

int main(void) {
	int count = 5;
	int block = 64;
	int grid = (count + block - 1) / block;

	// device Information
	int device = 0;
	hipSetDevice(device);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	printf("\ndevice #%d, %s\n", device, deviceProp.name);

	// static global memory
	float* value = (float*)malloc(sizeof(float));
	*value = 1.23;

	hipMemcpyToSymbol(HIP_SYMBOL(globalData), value, sizeof(float));
	changeGlobalData<<<1,1>>>();
	hipMemcpyFromSymbol(value, HIP_SYMBOL(globalData), sizeof(float));

	printf("static global memory test: %f\n", *value);

	// pinned memory
	int* pinned_array;
	hipHostAlloc((int**)&pinned_array, count* sizeof(int), hipHostMallocDefault);

	for (int x = 0; x < count; x++) {
		pinned_array[x] = x;
	}

	updateArray<<<grid,block>>>(pinned_array, count);
	hipDeviceSynchronize();

	printf("pinned memory test: ");
	printArray(pinned_array, count);

	hipHostFree(pinned_array);

	// zero-copy memory (with UVA)
	int* zero_copy_array;
	hipHostAlloc((int**)&zero_copy_array, count*sizeof(int), hipHostMallocMapped);
	for (int x = 0; x < count; x++) {
		zero_copy_array[x] = count - x;
	}
	updateArray<<<grid,block>>>(zero_copy_array, count);
	hipDeviceSynchronize();

	printf("zero-copy memory test: ");
	printArray(zero_copy_array, count);
	hipHostFree(zero_copy_array);

	// managed memory
	int* managed_memory;
	hipMallocManaged((int**)&managed_memory, count*sizeof(int));
	for (int x = 0; x < count; x++) {
		managed_memory[x] = x * 2;
	}
	updateArray<<<grid,block>>>(managed_memory, count);
	hipDeviceSynchronize();

	printf("managed memory: ");
	printArray(managed_memory, count);
	hipFree(managed_memory);

	// general device memory
	int* host_memory = (int*)malloc(count*sizeof(int));
	for (int x = 0; x < count; x++) {
		host_memory[x] = x * x + 1;
	}

	int* device_memory;
	hipMalloc((int**)&device_memory, count*sizeof(int));
	hipMemcpy(device_memory, host_memory, count*sizeof(int), hipMemcpyHostToDevice);
	printf("general device memory: ");
	printDeviceMemoryArray<<<grid,block>>>(device_memory, count);
	hipDeviceSynchronize();
	printf("\n");
	hipFree(device_memory);
	free(host_memory);

	hipDeviceReset();
	return 0;
}