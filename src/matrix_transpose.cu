
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void transposeReadRow(float* out, float* in, int rowCount, int columnCount) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (idx < columnCount && idy < rowCount) {
		out[idx*columnCount+idy] = in[idy*rowCount+idx];
	}
}

__global__
void transposeReadColumn(float* out, float* in, int rowCount, int columnCount) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (idx < columnCount && idy < rowCount) {
		out[idy*rowCount+idx] = in[idx*columnCount+idy];
	}
}

__global__
void transposeReadRowUnwrap8(float* out, float* in, int rowCount, int columnCount) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

	int x = idy*rowCount+idx;
	int y = idx*columnCount+idx;

	if (idx + 7 * blockDim.x < columnCount && idy < rowCount) {
		out[y] = in[x];
		out[y+rowCount*blockDim.x] = in[x+blockDim.x];
		out[y+2*rowCount*blockDim.x] = in[x+2*blockDim.x];
		out[y+3*rowCount*blockDim.x] = in[x+3*blockDim.x];
		out[y+4*rowCount*blockDim.x] = in[x+4*blockDim.x];
		out[y+5*rowCount*blockDim.x] = in[x+5*blockDim.x];
		out[y+6*rowCount*blockDim.x] = in[x+6*blockDim.x];
		out[y+7*rowCount*blockDim.x] = in[x+7*blockDim.x];

	}
}

__global__
void transposeReadColumnUnwrap8(float* out, float* in, int rowCount, int columnCount) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

	int x = idy*rowCount+idx;
	int y = idx*columnCount+idx;

	if (idx + 7 * blockDim.x < columnCount && idy < rowCount) {
		out[x] = in[y];
		out[x+blockDim.x] = in[y+rowCount*blockDim.x];
		out[x+2*blockDim.x] = in[y+2*rowCount*blockDim.x];
		out[x+3*blockDim.x] = in[y+3*rowCount*blockDim.x];
		out[x+4*blockDim.x] = in[y+4*rowCount*blockDim.x];
		out[x+5*blockDim.x] = in[y+5*rowCount*blockDim.x];
		out[x+6*blockDim.x] = in[y+6*rowCount*blockDim.x];
		out[x+7*blockDim.x] = in[y+7*rowCount*blockDim.x];
	}
}

void printMatrix(float* matrix, int rowCount, int columnCount) {
	for (int y = 0; y < columnCount; y++) {
		for (int x = 0; x < rowCount; x++) {
			printf("%-3d ", int(matrix[y*rowCount+x]));
		}
		printf("\n");
	}
}

int main(void) {
	int rowCount = 16;
	int columnCount = 16;

	dim3 block(32,32);
	dim3 grid((columnCount+block.x-1)/block.x, (rowCount+block.y-1)/block.y);
	dim3 gridUnwrap8((columnCount+(block.x/8)-1)/(block.x/8), (rowCount+block.y-1)/block.y);

	float* h_matrix = (float*)malloc(rowCount*columnCount*sizeof(float));
	for (int x = 0; x < rowCount * columnCount; x++) { h_matrix[x] = x; }
	float* h_transpose_matrix = (float*)malloc(rowCount*columnCount*sizeof(float));

	float* d_matrix;
	hipMalloc((float**)&d_matrix, rowCount*columnCount*sizeof(float));
	hipMemcpy(d_matrix, h_matrix, rowCount*columnCount*sizeof(float), hipMemcpyHostToDevice);

	float* d_transpose_matrix;
	hipMalloc((float**)&d_transpose_matrix, rowCount*columnCount*sizeof(float));

	transposeReadRow<<<grid,block>>>(d_transpose_matrix, d_matrix, rowCount, columnCount);
	hipDeviceSynchronize();
	hipMemcpy(h_transpose_matrix, d_transpose_matrix, rowCount*columnCount*sizeof(float), hipMemcpyDeviceToHost);

	transposeReadColumn<<<grid,block>>>(d_transpose_matrix, d_matrix, rowCount, columnCount);
	hipDeviceSynchronize();
	hipMemcpy(h_transpose_matrix, d_transpose_matrix, rowCount*columnCount*sizeof(float), hipMemcpyDeviceToHost);

	transposeReadRowUnwrap8<<<gridUnwrap8,block>>>(d_transpose_matrix, d_matrix, rowCount, columnCount);
	hipDeviceSynchronize();
	hipMemcpy(h_transpose_matrix, d_transpose_matrix, rowCount*columnCount*sizeof(float), hipMemcpyDeviceToHost);

	transposeReadColumnUnwrap8<<<gridUnwrap8,block>>>(d_transpose_matrix, d_matrix, rowCount, columnCount);
	hipDeviceSynchronize();
	hipMemcpy(h_transpose_matrix, d_transpose_matrix, rowCount*columnCount*sizeof(float), hipMemcpyDeviceToHost);

	printMatrix(h_transpose_matrix, rowCount, columnCount);

	hipFree(d_matrix);
	hipFree(d_transpose_matrix);
	free(h_matrix);
	free(h_transpose_matrix);
	hipDeviceReset();

	return 0;
}