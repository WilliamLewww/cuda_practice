
#include <hip/hip_runtime.h>
#include <stdio.h>
#define ARRAY_COUNT 10

__shared__ float file_shared_array_static[ARRAY_COUNT];
extern __shared__ float file_shared_array_dynamic[];

__global__
void generateArrayStatic(float* out) {
	__shared__ float array[ARRAY_COUNT];

	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < ARRAY_COUNT) {
		array[idx] = idx;
		out[idx] = array[idx];
	}
}

__global__
void generateArrayDynamic(float* out) {
	extern __shared__ float array[];

	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < ARRAY_COUNT) {
		array[idx] = idx;
		out[idx] = array[idx];
	}
}

__global__
void generateArrayFileStatic(float* out) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < ARRAY_COUNT) {
		file_shared_array_static[idx] = idx;
		out[idx] = file_shared_array_static[idx];
	}
}

__global__
void generateArrayFileDynamic(float* out) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < ARRAY_COUNT) {
		file_shared_array_dynamic[idx] = idx;
		out[idx] = file_shared_array_dynamic[idx];
	}
}

void printArray(float* array) {
	for (int x = 0; x < ARRAY_COUNT; x++) {
		printf("%d ", int(array[x]));
	}
	printf("\n");
}

int main(void) {
	printf("\n");
	dim3 block(32);
	dim3 grid((ARRAY_COUNT+block.x-1)/block.x);

	float* host_array = (float*)malloc(ARRAY_COUNT*sizeof(float));

	float* device_array;
	hipMalloc((float**)&device_array, ARRAY_COUNT*sizeof(float));

	generateArrayStatic<<<grid,block>>>(device_array);
	hipDeviceSynchronize();
	hipMemcpy(host_array, device_array, ARRAY_COUNT*sizeof(float), hipMemcpyDeviceToHost);
	printf("%-30s", "generateArrayStatic: ");
	printArray(host_array);

	generateArrayDynamic<<<grid,block,ARRAY_COUNT*sizeof(float)>>>(device_array);
	hipDeviceSynchronize();
	hipMemcpy(host_array, device_array, ARRAY_COUNT*sizeof(float), hipMemcpyDeviceToHost);
	printf("%-30s", "generateArrayDynamic: ");
	printArray(host_array);

	generateArrayFileStatic<<<grid,block>>>(device_array);
	hipDeviceSynchronize();
	hipMemcpy(host_array, device_array, ARRAY_COUNT*sizeof(float), hipMemcpyDeviceToHost);
	printf("%-30s", "generateArrayFileStatic: ");
	printArray(host_array);

	generateArrayFileDynamic<<<grid,block,ARRAY_COUNT*sizeof(float)>>>(device_array);
	hipDeviceSynchronize();
	hipMemcpy(host_array, device_array, ARRAY_COUNT*sizeof(float), hipMemcpyDeviceToHost);
	printf("%-30s", "generateArrayFileDynamic: ");
	printArray(host_array);

	free(host_array);
	hipFree(device_array);
	hipDeviceReset();
	printf("\n");
	return 0;
}