
#include <hip/hip_runtime.h>
#include <stdio.h>

#define OUTER_LOOP_COUNT 1000
#define INNER_LOOP_COUNT 1000

__global__
void warmUp(int* out, int* in, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size) return;
  
  out[idx] = in[idx];
}

__global__
void branchKernel(int* out, int* in, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size) return;

  int data = in[idx];

  for (int x = 0; x < OUTER_LOOP_COUNT; x++) {
    for (int y = 0; y < INNER_LOOP_COUNT; y++) {

      if (x % 2 == 0) { out[idx] += data; }
      if (y % 2 == 0) { out[idx] += data; }
      if ((x + y) % 2 == 0) { out[idx] += data; }

    }
  }
}

__global__
void calculateKernel(int* out, int* in, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size) return;

  int data = in[idx];

  for (int x = 0; x < OUTER_LOOP_COUNT; x++) {
    for (int y = 0; y < INNER_LOOP_COUNT; y++) {

      out[idx] += data * (x % 2 == 0);
      out[idx] += data * (y % 2 == 0);
      out[idx] += data * ((x + y) % 2 == 0);

    }
  }
}

int main(void) {
  int dataCount = 1000;
  int* h_data = (int*)malloc(dataCount*sizeof(int));

  dim3 block(32);
  dim3 grid((block.x + dataCount - 1) / block.x);

  int *d_data, *d_result;
  hipMalloc(&d_data, dataCount*sizeof(int));
  hipMalloc(&d_result, dataCount*sizeof(int));
  hipMemcpy(d_data, h_data, dataCount*sizeof(int), hipMemcpyHostToDevice);

  warmUp<<<grid, block>>>(d_result, d_data, dataCount);
  hipDeviceSynchronize();

  branchKernel<<<grid, block>>>(d_result, d_data, dataCount);
  hipDeviceSynchronize();

  calculateKernel<<<grid, block>>>(d_result, d_data, dataCount);
  hipDeviceSynchronize();

  hipFree(d_result);
  hipFree(d_data);
  free(h_data);
  
  return 0;
}