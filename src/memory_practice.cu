#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float globalData;

__global__ 
void changeGlobalData() {
	globalData += 1.00f;
}

int main(void) {
	float* value = (float*)malloc(sizeof(float));
	*value = 1.23;
	
	hipMemcpyToSymbol(HIP_SYMBOL(globalData), value, sizeof(float));
	changeGlobalData<<<1,1>>>();
	hipMemcpyFromSymbol(value, HIP_SYMBOL(globalData), sizeof(float));

	hipDeviceReset();

	printf("%f\n", *value);

	return 0;
}