#include "hip/hip_runtime.h"
#include <stdio.h>
#include "analysis.h"

__device__ float globalData;

__global__ 
void changeGlobalData() {
	globalData += 1.00f;
}

__global__
void updateArray(int* array) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	array[idx] *= 2.00f;
}

void printArray(int* array, int count) {
	for (int x = 0; x < count; x++) {
		printf("%d ", array[x]);
	}
	printf("\n");
}

int main(void) {
	int count = 5;
	int block = 64;
	int grid = (count + block - 1) / block;

	// device Information
	int device = 0;
	hipSetDevice(device);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	printf("\ndevice #%d, %s\n", device, deviceProp.name);

	// static global memory
	float* value = (float*)malloc(sizeof(float));
	*value = 1.23;

	hipMemcpyToSymbol(HIP_SYMBOL(globalData), value, sizeof(float));
	changeGlobalData<<<1,1>>>();
	hipMemcpyFromSymbol(value, HIP_SYMBOL(globalData), sizeof(float));

	printf("static global memory test: %f\n", *value);

	// pinned memory
	int* pinned_array;
	hipHostAlloc((int**)&pinned_array, count* sizeof(int), hipHostMallocDefault);

	for (int x = 0; x < count; x++) {
		pinned_array[x] = x;
	}

	updateArray<<<grid,block>>>(pinned_array);
	hipDeviceSynchronize();

	printf("pinned memory test: ");
	printArray(pinned_array, count);

	hipHostFree(pinned_array);

	// zero-copy memory (with UVA)
	int* zero_copy_array;
	hipHostAlloc((int**)&zero_copy_array, count*sizeof(int), hipHostMallocMapped);
	for (int x = 0; x < count; x++) {
		zero_copy_array[x] = count - x;
	}
	updateArray<<<grid,block>>>(zero_copy_array);
	hipDeviceSynchronize();

	printf("zero-copy memory test: ");
	printArray(zero_copy_array, count);
	hipHostFree(zero_copy_array);

	// managed memory
	int* managed_memory;
	hipMallocManaged((int**)&managed_memory, count*sizeof(int));
	for (int x = 0; x < count; x++) {
		managed_memory[x] = x * 2;
	}
	updateArray<<<grid,block>>>(managed_memory);
	hipDeviceSynchronize();

	printf("managed memory: ");
	printArray(managed_memory, count);
	hipFree(managed_memory);

	hipDeviceReset();
	return 0;
}