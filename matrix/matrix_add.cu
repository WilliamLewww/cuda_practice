#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <time.h>
#include <cmath>

void printMatrix(float* matrix, int rowCount, int columnCount) {
}

void initializeMatrix(float* matrix, int rowCount, int columnCount) {
	for (int y = 0; y < rowCount; y++) {
		for (int x = 0; x < columnCount; x++) {
			matrix[(y * columnCount) + x] = rand() % 25;
		}
	}
}

__global__
void addMatrix(float* out, float* a, float* b) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	out[x] = a[x] + b[x];
}

int main(void) {
	srand(time(NULL));

	int matrixRowCount = 4;
	int matrixColumnCount = 4;
	int memorySize = matrixRowCount*matrixColumnCount*sizeof(float);

	float* h_a = (float*)malloc(memorySize);
	float* h_b = (float*)malloc(memorySize);
	float* h_c = (float*)malloc(memorySize);

	initializeMatrix(h_a, matrixRowCount, matrixColumnCount);
	initializeMatrix(h_b, matrixRowCount, matrixColumnCount);

	float *d_a, *d_b, *d_c;
	hipMalloc((float**)&d_a, memorySize);
	hipMalloc((float**)&d_b, memorySize);
	hipMalloc((float**)&d_c, memorySize);

	hipMemcpy(d_a, h_a, memorySize, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, memorySize, hipMemcpyHostToDevice);

	dim3 block(256);
	dim3 grid((matrixSize + block.x - 1) / block.x);
	addMatrix<<<grid, block>>>(d_c, d_a, d_b);

	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, memorySize, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printMatrix(h_a, matrixSize);
	printMatrix(h_b, matrixSize);
	printMatrix(h_c, matrixSize);

	hipDeviceReset();
	return 0;
}