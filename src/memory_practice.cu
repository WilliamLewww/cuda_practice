#include "hip/hip_runtime.h"
#include <stdio.h>
#include "analysis.h"

__device__ float globalData;

__global__ 
void changeGlobalData() {
	globalData += 1.00f;
}

__global__
void updateArray(int* array) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	array[idx] *= 2.00f;
}

int main(void) {
	int count = 5;
	int block = 64;
	int grid = (count + block - 1) / block;

	// device Information
	int device = 0;
	hipSetDevice(device);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	printf("\ndevice #%d, %s\n", device, deviceProp.name);

	// global memory
	float* value = (float*)malloc(sizeof(float));
	*value = 1.23;

	hipMemcpyToSymbol(HIP_SYMBOL(globalData), value, sizeof(float));
	changeGlobalData<<<1,1>>>();
	hipMemcpyFromSymbol(value, HIP_SYMBOL(globalData), sizeof(float));

	printf("static global memory test: %f\n", *value);

	// pinned memory
	int* pinned_array;
	hipHostAlloc((int**)&pinned_array, count* sizeof(int), hipHostMallocDefault);

	for (int x = 0; x < count; x++) {
		pinned_array[x] = x;
	}

	updateArray<<<grid,block>>>(pinned_array);
	hipDeviceSynchronize();

	printf("pinned memory test: ");
	for (int x = 0; x < count; x++) {
		printf("%d ", pinned_array[x]);
	}
	printf("\n");

	hipHostFree(pinned_array);

	// zero-copy memory
	int* zero_copy_array;
	hipHostAlloc((int**)&zero_copy_array, count*sizeof(int), hipHostMallocMapped);
	for (int x = 0; x < count; x++) {
		zero_copy_array[x] = count - x;
	}
	updateArray<<<grid,block>>>(zero_copy_array);
	hipDeviceSynchronize();

	printf("zero-copy memory test: ");
	for (int x = 0; x < count; x++) {
		printf("%d ", zero_copy_array[x]);
	}
	printf("\n");
	hipHostFree(zero_copy_array);

	hipDeviceReset();
	return 0;
}