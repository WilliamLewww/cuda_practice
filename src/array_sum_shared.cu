
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void sumUnrollGlobal(float* out, float* in, int count) {
	float* local_array = in + (blockIdx.x * blockDim.x);

	if (blockDim.x >= 1024 && threadIdx.x < 512) { local_array[threadIdx.x] += local_array[threadIdx.x + 512]; }
	__syncthreads();
	if (blockDim.x >= 512 && threadIdx.x < 256) { local_array[threadIdx.x] += local_array[threadIdx.x + 256]; }
	__syncthreads();
	if (blockDim.x >= 256 && threadIdx.x < 128) { local_array[threadIdx.x] += local_array[threadIdx.x + 128]; }
	__syncthreads();
	if (blockDim.x >= 128 && threadIdx.x < 64) { local_array[threadIdx.x] += local_array[threadIdx.x + 64]; }
	__syncthreads();

	if (threadIdx.x < 32) {
		volatile float* v_local_array = local_array;
		v_local_array[threadIdx.x] += v_local_array[threadIdx.x + 32];
		v_local_array[threadIdx.x] += v_local_array[threadIdx.x + 16];
		v_local_array[threadIdx.x] += v_local_array[threadIdx.x + 8];
		v_local_array[threadIdx.x] += v_local_array[threadIdx.x + 4];
		v_local_array[threadIdx.x] += v_local_array[threadIdx.x + 2];
		v_local_array[threadIdx.x] += v_local_array[threadIdx.x + 1];
	}

	if (threadIdx.x == 0) { out[blockIdx.x] = local_array[0]; }
}

int main(void) {
	int count = 1 << 16;
	dim3 block(32);
	dim3 grid((count + block.x - 1) / block.x);

	float* host_array = (float*)malloc(count*sizeof(float));
	for (int x = 0; x < count; x++) { host_array[x] = x; }
	float* host_result_array = (float*)malloc(grid.x*sizeof(float));

	float *device_array, *device_result_array;
	hipMalloc((float**)&device_array, count*sizeof(float));
	hipMemcpy(device_array, host_array, count*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((float**)&device_result_array, grid.x*sizeof(float));

	sumUnrollGlobal<<<grid,block>>>(device_result_array, device_array, count);
	hipDeviceSynchronize();

	hipMemcpy(host_result_array, device_result_array, grid.x*sizeof(float), hipMemcpyDeviceToHost);

	free(host_array);
	free(host_result_array);
	hipFree(device_array);
	hipFree(device_result_array);
	hipDeviceReset();
	return 0;
}