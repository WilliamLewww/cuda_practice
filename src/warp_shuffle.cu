
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void shuffleUp(float* out, float* in, int count) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx > count) return;

	float local_value = in[idx];
	local_value = __shfl_up_sync(0xffffffff, local_value, 1);

	out[idx] = local_value;
}

__global__
void shuffleDown(float* out, float* in, int count) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx > count) return;

	float local_value = in[idx];
	local_value = __shfl_down_sync(0xffffffff, local_value, 1);

	out[idx] = local_value;
}

__global__
void shuffleButterfly(float* out, float* in, int count) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx > count) return;

	float local_value = in[idx];
	local_value = __shfl_xor_sync(0xffffffff, local_value, 1);

	out[idx] = local_value;
}

int main(void) {
	printf("\n");

	int count = 32;
	dim3 block(8);
	dim3 grid((count + block.x - 1) / block.x);

	float* h_array = (float*)malloc(count*sizeof(float));
	for (int x = 0; x < count; x++) { h_array[x] = x; }
	float* h_result_array = (float*)malloc(count*sizeof(float));

	float *d_array, *d_result_array;
	hipMalloc((float**)&d_array, count*sizeof(float));
	hipMemcpy(d_array, h_array, count*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((float**)&d_result_array, count*sizeof(float));

	shuffleUp<<<grid, block>>>(d_result_array, d_array, count);
	hipDeviceSynchronize();
	hipMemcpy(h_result_array, d_result_array, count*sizeof(float), hipMemcpyDeviceToHost);
	for (int x = 0; x < count; x++) { printf("%d ", int(h_result_array[x])); }
	printf("\n");

	shuffleDown<<<grid, block>>>(d_result_array, d_array, count);
	hipDeviceSynchronize();
	hipMemcpy(h_result_array, d_result_array, count*sizeof(float), hipMemcpyDeviceToHost);
	for (int x = 0; x < count; x++) { printf("%d ", int(h_result_array[x])); }
	printf("\n");

	shuffleButterfly<<<grid, block>>>(d_result_array, d_array, count);
	hipDeviceSynchronize();
	hipMemcpy(h_result_array, d_result_array, count*sizeof(float), hipMemcpyDeviceToHost);
	for (int x = 0; x < count; x++) { printf("%d ", int(h_result_array[x])); }
	printf("\n");

	free(h_array);
	free(h_result_array);
	hipFree(d_array);
	hipFree(d_result_array);
	hipDeviceReset();

	printf("\n");
	return 0;
}