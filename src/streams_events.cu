
#include <hip/hip_runtime.h>
#include <stdio.h>
#define STREAM_COUNT 3

__global__
void createIncrementingArray(float* out, int count) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx < count) {
		out[idx] = idx;
	}
}

void warmUp(int count, dim3 block, dim3 grid) {
	float* result;
	hipMalloc((float**)&result, count*sizeof(float));

	createIncrementingArray<<<grid, block>>>(result, count);
	hipFree(result);
}

void createArraysNullStream(int count, dim3 block, dim3 grid) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	float *d_result_first, *d_result_second, *d_result_third;
	hipMalloc((float**)&d_result_first, count*sizeof(float));
	hipMalloc((float**)&d_result_second, count*sizeof(float));
	hipMalloc((float**)&d_result_third, count*sizeof(float));

	for (int x = 0; x < STREAM_COUNT; x++) {
		createIncrementingArray<<<grid, block>>>(d_result_first, count);
		createIncrementingArray<<<grid, block>>>(d_result_second, count);
		createIncrementingArray<<<grid, block>>>(d_result_third, count);
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float time;
	hipEventElapsedTime(&time, start, stop);

	printf("%-30s%f\n", "Null Stream: ", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_result_first);
	hipFree(d_result_second);
	hipFree(d_result_third);
}

void createArrayNonNullStream(int count, dim3 block, dim3 grid) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	float *d_result_first, *d_result_second, *d_result_third;
	hipMalloc((float**)&d_result_first, count*sizeof(float));
	hipMalloc((float**)&d_result_second, count*sizeof(float));
	hipMalloc((float**)&d_result_third, count*sizeof(float));

	int streamCount = STREAM_COUNT;
	hipStream_t* streams = (hipStream_t*)malloc(streamCount*sizeof(hipStream_t));
	for (int x = 0; x < streamCount; x++) {
		hipStreamCreate(&streams[x]);
	}

	for (int x = 0; x < streamCount; x++) {
		createIncrementingArray<<<grid, block, 0, streams[x]>>>(d_result_first, count);
		createIncrementingArray<<<grid, block, 0, streams[x]>>>(d_result_first, count);
		createIncrementingArray<<<grid, block, 0, streams[x]>>>(d_result_first, count);
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float time;
	hipEventElapsedTime(&time, start, stop);

	printf("%-30s%f\n", "Non-Null Stream: ", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_result_first);
	hipFree(d_result_second);
	hipFree(d_result_third);
}

int main(void) {
	printf("\n");
	int count = 1 << 16;
	dim3 block(32);
	dim3 grid((count + block.x - 1) / block.x);

	warmUp(count, block, grid);
	createArraysNullStream(count, block, grid);
	createArrayNonNullStream(count, block, grid);
	hipDeviceReset();

	printf("\n");
}