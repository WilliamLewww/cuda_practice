
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float globalData;

__global__ 
void changeGlobalData() {
	globalData += 1.00f;
}

int main(void) {
	int device = 0;
	hipSetDevice(device);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	printf("\nDevice #%d, %s\n", device, deviceProp.name);

	float* value = (float*)malloc(sizeof(float));
	*value = 1.23;

	hipMemcpyToSymbol(HIP_SYMBOL(globalData), value, sizeof(float));
	changeGlobalData<<<1,1>>>();
	hipMemcpyFromSymbol(value, HIP_SYMBOL(globalData), sizeof(float));

	hipDeviceReset();

	printf("%f\n", *value);

	return 0;
}