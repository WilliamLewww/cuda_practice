
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void reduceNeighboredLess(int* g_i, int* g_o, int count) {
	
}

int main(void) {
	int count = 1 << 16;
	int size = count * sizeof(int);

	int* h_i = (int*)malloc(size);
	int* h_o = (int*)malloc(size);

	for (int x = 0; x < count; x++) { h_i[x] = x; }

	int *d_i, *d_o;
	hipMalloc((float**)&d_i, size);
	hipMalloc((float**)&d_o, size);

	hipMemcpy(d_i, h_i, size, hipMemcpyHostToDevice);

	dim3 block = 128;
	dim3 grid = ((count + block.x - 1) / block.x);
	reduceNeighboredLess<<<grid, block>>>(d_i, d_o, count);
	hipDeviceSynchronize();

	hipMemcpy(h_o, d_o, size, hipMemcpyDeviceToHost);
	hipFree(d_i);
	hipFree(d_o);

	int sum = 0;
	for (int x = 0; x < grid.x + 5; x++) {
		sum += h_o[x];
	}

	printf("\ntotal sum (0 to %d): %d\n\n", count, sum);

	free(h_i);
	free(h_o);
	hipDeviceReset();

	return 0;
}