
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void shuffleUp(float* out, float* in, int count) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx > count) return;

	float local_value = in[idx];
	local_value = __shfl_up_sync(0xffffffff, local_value, 1);

	out[idx] = local_value;
}

int main(void) {
	int count = 32;
	dim3 block(32);
	dim3 grid((count + block.x - 1) / block.x);

	float* h_array = (float*)malloc(count*sizeof(float));
	for (int x = 0; x < count; x++) { h_array[x] = x; }
	float* h_result_array = (float*)malloc(count*sizeof(float));

	float *d_array, *d_result_array;
	hipMalloc((float**)&d_array, count*sizeof(float));
	hipMemcpy(d_array, h_array, count*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((float**)&d_result_array, count*sizeof(float));

	shuffleUp<<<grid, block>>>(d_result_array, d_array, count);
	hipDeviceSynchronize();
	hipMemcpy(h_result_array, d_result_array, count*sizeof(float), hipMemcpyDeviceToHost);

	free(h_array);
	free(h_result_array);
	hipFree(d_array);
	hipFree(d_result_array);
	hipDeviceReset();
	return 0;
}